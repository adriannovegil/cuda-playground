/**
 * Lecture 4
 *
 * Programación de GPUs (General Purpose Computation on Graphics Processing
 * Unit)
 *
 * PCR en GPU
 * Parámetros opcionales (en este orden): sumavectores #rep #n #blk
 * #rep: número de repetiones
 * #n: número de elementos en cada vector
 * #blk: hilos por bloque CUDA
 */
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

const int N = 1024;                  // Número predeterm. de elementos en los vectores
const int CUDA_BLK = 16;             // Tamaño predeterm. de bloque de hilos ƒCUDA
const int NUMBER_OF_SYSTEMS = 32760; // Cantidad de sistemas a calcular

/**
 * Para medir el tiempo transcurrido (elapsed time):
 *
 * resnfo: tipo de dato definido para abstraer la métrica de recursos a usar
 * timenfo: tipo de dato definido para abstraer la métrica de tiempo a usar
 *
 * timestamp: abstrae función usada para tomar las muestras del tiempo transcurrido
 *
 * printtime: abstrae función usada para imprimir el tiempo transcurrido
 *
 * void myElapsedtime(resnfo start, resnfo end, timenfo *t): función para obtener
 * el tiempo transcurrido entre dos medidas
 */
#ifdef _noWALL_
typedef struct rusage resnfo;
typedef struct _timenfo
{
    double time;
    double systime;
} timenfo;
#define timestamp(sample) getrusage(RUSAGE_SELF, (sample))
#define printtime(t) printf("%15f s (%f user + %f sys) ", \
                            t.time + t.systime, t.time, t.systime);
#else
typedef struct timeval resnfo;
typedef double timenfo;
#define timestamp(sample) gettimeofday((sample), 0)
#define printtime(t) printf("%15f s ", t);
#endif

void myElapsedtime(const resnfo start, const resnfo end, timenfo *const t)
{
#ifdef _noWALL_
    t->time = (end.ru_utime.tv_sec + (end.ru_utime.tv_usec * 1E-6)) - (start.ru_utime.tv_sec + (start.ru_utime.tv_usec * 1E-6));
    t->systime = (end.ru_stime.tv_sec + (end.ru_stime.tv_usec * 1E-6)) - (start.ru_stime.tv_sec + (start.ru_stime.tv_usec * 1E-6));
#else
    *t = (end.tv_sec + (end.tv_usec * 1E-6)) - (start.tv_sec + (start.tv_usec * 1E-6));
#endif /*_noWALL_*/
}

/**
 * Prints the values of the array to the screen
 */
void print_array(float *array, const unsigned int m)
{
    unsigned int i;
    for (i = 0; i < m; i++)
    {
        printf("%f ", array[i]);
    }
}

/**
 * Prints the values of the matrix to the screen
 */
void print_matrix(float *matrix, const unsigned int m, const unsigned int n)
{
    unsigned int i, j;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", matrix[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

/**
 * Function that comprate the elements of two array's
 */
bool compare_array(float *a1, float *a2, const unsigned int m)
{
    for (int i = 0; i < m; i++)
        if (a1[i] != a2[i])
        {
            printf("Mismatch at index %d, was: %f, should be: %f\n", i, a1[i], a2[i]);
            return false;
        }
    return true;
}

/**
 * Función para inicializar los vectores que vamos a utilizar
 */
void systemInitialization(float A[], float B[], float C[], float D[], const unsigned int n)
{
    unsigned int i;

    A[0] = 0.0;
    B[0] = 2.0;
    C[0] = -1.0;
    D[0] = 1.0;

    for (i = 1; i < n - 1; i++)
    {
        A[i] = -1.0;
        B[i] = 2.0;
        C[i] = -1.0;
        D[i] = 0.0;
    }

    A[n - 1] = -1.0;
    B[n - 1] = 2.0;
    C[n - 1] = 0.0;
    D[n - 1] = 1.0;
}

/**
 * Función que inicializa la matriz de sistemas
 */
void systemsInitialization(float A[], float B[], float C[], float D[],
                           const unsigned int nSystems,
                           const unsigned int nElements)
{
    unsigned int system;
    for (system = 0; system < nSystems; system++)
    {
        systemInitialization(&A[(system * nElements)],
                             &B[(system * nElements)],
                             &C[(system * nElements)],
                             &D[(system * nElements)],
                             nElements);
    }
}

/**
 * Función que calcula el resutlado final del sistema
 */
void calculateResult(float X[], float Y[], float Z[], float W[], const unsigned int n)
{
    for (int j = 0; j < n / 2; j++)
    {
        float temp;
        temp = Y[j + n / 2] * Y[j] - Z[j] * X[j + n / 2];
        X[j] = (Y[j + n / 2] * W[j] - Z[j] * W[j + n / 2]) / temp;
        X[j + n / 2] = (W[j + n / 2] * Y[j] - W[j] * X[j + n / 2]) / temp;
    }
}

// CPU execution
// ============================================================================

/**
 *
 */
void pcr_cpu_kernel(float X[], float Y[], float Z[], float W[], const unsigned int n)
{
    unsigned int i, k;
    unsigned ln = floor(log2(float(n)));
    float alpha, gamma;

    unsigned int numBytes = n * sizeof(float);

    float *Xr = (float *)malloc(numBytes);
    float *Yr = (float *)malloc(numBytes);
    float *Zr = (float *)malloc(numBytes);
    float *Wr = (float *)malloc(numBytes);

    k = 1;
    for (i = 0; i < ln; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (j >= k)
            {
                if (j <= (n - k - 1))
                {
                    alpha = -X[j] / Y[j - k];
                    gamma = -Z[j] / Y[j + k];
                    Yr[j] = Y[j] + (alpha * Z[j - k] + gamma * X[j + k]);
                    Xr[j] = alpha * X[j - k];
                    Zr[j] = gamma * Z[j + k];
                    Wr[j] = W[j] + (alpha * W[j - k] + gamma * W[j + k]);
                }
                else
                {
                    alpha = -X[j] / Y[j - k];
                    Yr[j] = Y[j] + (alpha * Z[j - k]);
                    Xr[j] = alpha * X[j - k];
                    Zr[j] = 0;
                    Wr[j] = W[j] + (alpha * W[j - k]);
                }
            }
            else
            {
                gamma = -Z[j] / Y[j + k];
                Yr[j] = Y[j] + gamma * X[j + k];
                Xr[j] = 0;
                Zr[j] = gamma * Z[j + k];
                Wr[j] = W[j] + gamma * W[j + k];
            }
        }
        k = k << 1;
        for (int j = 0; j < n; j++)
        {
            X[j] = Xr[j];
            Y[j] = Yr[j];
            Z[j] = Zr[j];
            W[j] = Wr[j];
        }
    }

    calculateResult(X, Y, Z, W, n);

    for (int j = 0; j < n; j++)
    {
        printf(" \t %f  \n", X[j]);
    }
}

/**
 * Función PCR en la CPU
 */
void pcr_cpu(const unsigned int n)
{
    // Para medir tiempos
    resnfo start, end;
    timenfo time;

    unsigned int numBytes = n * sizeof(float);

    // Reservamos e inicializamos vectores
    timestamp(&start);
    float *h_Av = (float *)malloc(numBytes);
    float *h_Bv = (float *)malloc(numBytes);
    float *h_Cv = (float *)malloc(numBytes);
    float *h_Dv = (float *)malloc(numBytes);
    systemInitialization(h_Av, h_Bv, h_Cv, h_Dv, n);
    timestamp(&end);
    myElapsedtime(start, end, &time);
    printtime(time);
    printf(" -> Reservar e inicializar vectores CPU (%u)\n\n", n);

    // CPU execution
    timestamp(&start);
    pcr_cpu_kernel(h_Av, h_Bv, h_Cv, h_Dv, n);
    timestamp(&end);
    myElapsedtime(start, end, &time);
    printtime(time);
    printf(" -> PCR en la CPU\n\n");

    free(h_Av);
    free(h_Bv);
    free(h_Cv);
    free(h_Dv);
}

// GPU execution
// ============================================================================

/**
 * Kernel definition
 */
extern __shared__ float array[];
__global__ void pcr_gpu_kernel(float *X, float *Y, float *Z, float *W,
                               const unsigned int number_of_systems,
                               const unsigned int n)
{
    unsigned int i, k;
    unsigned ln = floor(log2(float(n)));
    float alpha, gamma;

    int global_pos = blockDim.y * blockIdx.y + threadIdx.y;
    int row = threadIdx.y;

    float *Xs = (float *)array;
    float *Ys = (float *)&Xs[number_of_systems * n];
    float *Zs = (float *)&Ys[number_of_systems * n];
    float *Ws = (float *)&Zs[number_of_systems * n];

    float Xr, Yr, Zr, Wr;

    if (global_pos < number_of_systems * n)
    {
        k = 1;
        for (i = 0; i < ln; i++)
        {
            Xs[threadIdx.y] = X[global_pos];
            Ys[threadIdx.y] = Y[global_pos];
            Zs[threadIdx.y] = Z[global_pos];
            Ws[threadIdx.y] = W[global_pos];
            // We synchronize threads to ensure the loading of the entire sub-array
            __syncthreads();

            // for (int j = 0; j < n; j++)
            //{
            if (row >= k)
            {
                if (row <= (n - k - 1))
                {
                    alpha = -Xs[row] / Ys[row - k];
                    gamma = -Zs[row] / Ys[row + k];
                    Yr = Ys[row] + (alpha * Zs[row - k] + gamma * Xs[row + k]);
                    Xr = alpha * Xs[row - k];
                    Zr = gamma * Zs[row + k];
                    Wr = Ws[row] + (alpha * Ws[row - k] + gamma * Ws[row + k]);
                }
                else
                {
                    alpha = -Xs[row] / Ys[row - k];
                    Yr = Ys[row] + (alpha * Zs[row - k]);
                    Xr = alpha * Xs[row - k];
                    Zr = 0;
                    Wr = Ws[row] + (alpha * Ws[row - k]);
                }
            }
            else
            {
                gamma = -Zs[row] / Ys[row + k];
                Yr = Ys[row] + gamma * Xs[row + k];
                Xr = 0;
                Zr = gamma * Zs[row + k];
                Wr = Ws[row] + gamma * Ws[row + k];
            }
            //}

            __syncthreads();

            k = k << 1;

            // for (int j = 0; j < n; j++)
            //{
            X[global_pos] = Xr;
            Y[global_pos] = Yr;
            Z[global_pos] = Zr;
            W[global_pos] = Wr;
            //}
        }
    }
}

/**
 * Función PCR en la GPU
 */
void pcr_gpu(const unsigned int number_of_systems,
             const unsigned int n,
             const unsigned int block_size)
{
    // Para medir tiempos
    resnfo startgpu, endgpu;
    timenfo timegpu;

    float *d_X, *d_Y, *d_Z, *d_W;

    // Número de bytes a reservar para nuestros vectores
    unsigned int numBytes = number_of_systems * n * sizeof(float);
    unsigned int systemsMatrixNumBytes = number_of_systems * n * sizeof(float);

    // Reservamos e inicializamos vectores
    timestamp(&startgpu);
    float *X = (float *)malloc(systemsMatrixNumBytes);
    float *Y = (float *)malloc(systemsMatrixNumBytes);
    float *Z = (float *)malloc(systemsMatrixNumBytes);
    float *W = (float *)malloc(systemsMatrixNumBytes);
    systemsInitialization(X, Y, Z, W, number_of_systems, n);
    hipMalloc(&d_X, numBytes);
    hipMalloc(&d_Y, numBytes);
    hipMalloc(&d_Z, numBytes);
    hipMalloc(&d_W, numBytes);
    hipMemcpy(d_X, X, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Z, Z, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, numBytes, hipMemcpyHostToDevice);
    timestamp(&endgpu);
    myElapsedtime(startgpu, endgpu, &timegpu);
    printtime(timegpu);
    printf(" -> Reservar e inicializar vectores GPU (%u)\n\n", n);

    // Launch kernel
    //  - threads_per_block: number of CUDA threads per grid block
    //	- blocks_in_grid   : number of blocks in grid
    //	(These are c structs with 3 member variables x, y, x)
    dim3 threads_per_block(1,
                           block_size,
                           1); // dim3 variable holds 3 dimensions
    dim3 blocks_in_grid(1,
                        number_of_systems,
                        // ceil(float(n) / threads_per_block.y),
                        1);
    unsigned int sharedSize = numBytes * 4;
    timestamp(&startgpu);
    pcr_gpu_kernel<<<blocks_in_grid, threads_per_block, sharedSize>>>(d_X, d_Y, d_Z, d_W, number_of_systems, n);
    hipDeviceSynchronize();
    timestamp(&endgpu);
    myElapsedtime(startgpu, endgpu, &timegpu);
    printtime(timegpu);
    printf(" -> PCR en la GPU\n\n");

    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    hipError_t cuErrSync = hipGetLastError();
    if (cuErrSync != hipSuccess)
    {
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync));
        exit(0);
    }

    // Check for errors on the GPU after control is returned to CPU
    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess)
    {
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync));
        exit(0);
    }

    // Copy data from device to CPU
    hipMemcpy(X, d_X, numBytes, hipMemcpyDeviceToHost);
    hipMemcpy(Y, d_Y, numBytes, hipMemcpyDeviceToHost);
    hipMemcpy(Z, d_Z, numBytes, hipMemcpyDeviceToHost);
    hipMemcpy(W, d_W, numBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < number_of_systems; i++)
    {
        calculateResult(&X[(i * n)],
                        &Y[(i * n)],
                        &Z[(i * n)],
                        &W[(i * n)],
                        n);
    }

    printf(" Av= [");
    print_matrix(X, number_of_systems, n);
    printf("]\n\n");

    // Free CPU and GPU memory
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Z);
    hipFree(d_W);

    free(X);
    free(Y);
    free(Z);
    free(W);
}

// Main program
// ============================================================================

/**
 * Función principal
 */
int main(int argc, char *argv[])
{
    // Read program arguments
    unsigned int n = (argc > 1) ? atoi(argv[1]) : N;
    unsigned int block_size = (argc > 2) ? atoi(argv[2]) : CUDA_BLK;
    unsigned int number_of_systems = (argc > 3) ? atoi(argv[3]) : NUMBER_OF_SYSTEMS;

    printf("--------------------------------\n");
    printf(" Parallel Cyclic Reduction (PCR)\n");
    printf("--------------------------------\n");

    // Llamada a la función d ejecución de la CPU
    pcr_cpu(n);

    // Llamada a la función d ejecución de la GPU
    pcr_gpu(number_of_systems, n, block_size);

    printf(" Number of systems         = %d\n", number_of_systems);
    printf(" System size               = %d\n", n);
    printf("--------------------------------\n");
    printf(" SUCCESS\n");
    printf("--------------------------------\n");

    return (0);
}
